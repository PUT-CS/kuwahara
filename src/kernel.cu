#include "hip/hip_runtime.h"
#include "pixel.cuh"
#include "quadrant.cuh"
#include "kernel.cuh"
#include "stdio.h"

__device__ void updateVariance(QuadrantData& quadrant, double newValue) {
    quadrant.count++;
    double delta = newValue - quadrant.varianceMean;
    quadrant.varianceMean += delta / quadrant.count;
    double delta2 = newValue - quadrant.varianceMean;
    quadrant.varianceM2 += delta * delta2;
}

__device__ double finalizeVariance(QuadrantData& quadrant) {
    if (quadrant.count < 2) {
        return 255; // return a large value to indicate that the variance is not valid
    }
    return quadrant.varianceM2 / quadrant.count; // sample variance
}

/// A pixel belongs to two quadrants at the same time.
/// Fills the 2 int indexes with the quadrant values, else -1.
/// The first field is always set. The second field is set only if the pixel belongs to two quadrants.
__device__ QuadrantResult checkQuadrant(int i, int j)
{
    if (i < 0 && j < 0) {
        return { TOP_LEFT, NONE };
    } else if (i > 0 && j < 0) {
        return { TOP_RIGHT, NONE };
    } else if (i < 0 && j > 0) {
        return { BOTTOM_LEFT, NONE };
    } else if (i > 0 && j > 0) {
        return { BOTTOM_RIGHT, NONE };
    } else if (i == 0) {
        if (j < 0) {
            return { TOP_LEFT, TOP_RIGHT };
        } else if (j > 0) {
            return { BOTTOM_LEFT, BOTTOM_RIGHT };
        }
    } else if (j == 0) {
        if (i < 0) {
            return { TOP_LEFT, BOTTOM_LEFT };
        } else if (i > 0) {
            return { TOP_RIGHT, BOTTOM_RIGHT };
        }
    }
    return {NONE, NONE};
}

/// calculate the luminosity of a BGR pixel
__device__ double luminosity(const BGRPixel &pixel) {
    return 0.299 * pixel.pixel.r + 0.587 * pixel.pixel.g + 0.114 * pixel.pixel.b;
}

__device__ bool pixelInBounds(int x, int y, int sizeX, int sizeY) {
    return x >= 0 && x < sizeX && y >= 0 && y < sizeY;
}

__device__ void countPixel(QuadrantData &quadrantData, BGRPixel &pixel) {
    quadrantData.bSum += pixel.data[0];
    quadrantData.gSum += pixel.data[1];
    quadrantData.rSum += pixel.data[2];
    updateVariance(quadrantData, pixel.pixel.luminosity); // count is updated here
}

__device__ double findIndexOfMinStdDev(QuadrantData quadrants[4]) {
    int minIdx = -1;
    double minStdDev = 255;

    for (int i = 0; i < 4; i++) {
        if (quadrants[i].count == 0) {
            continue;
        }
        double currStdDev = std::sqrt(finalizeVariance(quadrants[i]));
        if (currStdDev < minStdDev) { // new minimum
            minStdDev = currStdDev;
            minIdx = i;
        }
    }
    return minIdx;
}

__device__ BGRPixel avgOfQuadrant(QuadrantData &quadrant) {
    return {static_cast<uchar>(quadrant.bSum / quadrant.count),
            static_cast<uchar>(quadrant.gSum / quadrant.count),
            static_cast<uchar>(quadrant.rSum / quadrant.count), 0};
}

__device__ void processQuadrants(QuadrantData quadrants[4], BGRPixel *image, int x, int y,
                                        int sizeX, int sizeY, int quadrantSize) {
    for (int i = -quadrantSize + 1; i < quadrantSize; i++) {
        for (int j = -quadrantSize + 1; j < quadrantSize; j++) {
            // check if the pixel is within the bounds of the image
            int pixelX = x + i, pixelY = y + j;
            if (!pixelInBounds(pixelX, pixelY, sizeX, sizeY) || i == 0 || j == 0)
                continue;
            auto &neighbourPixel = image[pixelY * sizeX + pixelX];

            // calculate luminosity of the rbg pixel to avod the problem
            // described here
            // https://en.wikipedia.org/wiki/Kuwahara_filter#Color_images
            auto pixelLuminosity = static_cast<uchar>(luminosity(neighbourPixel));
            BGRPixel bgrPixel = {static_cast<uchar>(neighbourPixel.data[0]),
                                 static_cast<uchar>(neighbourPixel.data[1]),
                                 static_cast<uchar>(neighbourPixel.data[2]), pixelLuminosity};

            // check which quadrants the pixel belongs to
            auto quadrantResult = checkQuadrant(i, j);
            
            // add the pixel to quadrant arrays
            countPixel(quadrants[quadrantResult.q1], bgrPixel);
            // if the pixel belongs to two quadrants, add it to the second one
            if (quadrantResult.q2 != NONE) {
                countPixel(quadrants[quadrantResult.q2], bgrPixel);
            }
        }
    }
}

__global__ void kuwahara(BGRPixel *image, BGRPixel *outputImage, int sizeX, int sizeY, int quadrantSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= sizeX || y >= sizeY) {
        return;
    }
    
    QuadrantData quadrants[4] = {};
    processQuadrants(quadrants, image, x, y, sizeX, sizeY, quadrantSize);
    int minIdx = findIndexOfMinStdDev(quadrants);
    outputImage[y * sizeX + x] = avgOfQuadrant(quadrants[minIdx]);
}

namespace KernelWrapper {
  void launchKuwaharaKernel(BGRPixel *image, BGRPixel *outputImage, int sizeX, int sizeY, int quadrantSize) {
    const int blockX = 16;
    const int blockY = 16;
    dim3 numberOfBlocks(sizeX / blockX + 1, sizeY / blockY + 1);
    dim3 numberOfThreads(blockX, blockY);
    kuwahara<<<numberOfBlocks, numberOfThreads>>>(image, outputImage, sizeX, sizeY, quadrantSize);
  }
}